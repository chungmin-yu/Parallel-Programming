#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BLOCK_SIZE 25
#define GROUP_SIZE 4

__global__ void mandelKernel(int* data, float lowerX, float lowerY, float stepX, float stepY, int maxIteration, int pitch, int scale) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
   
    
    int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * scale;
    int thisY = (blockIdx.y * blockDim.y + threadIdx.y) * scale;

    for (int a = 0; a < scale; a++){
	for (int b = 0; b < scale; b++){
    	    float c_re = lowerX + (thisX + b) * stepX;
    	    float c_im = lowerY + (thisY + a) * stepY;
    	    float z_re = c_re, z_im = c_im;
     
	    int i;
    	    for (i = 0; i < maxIteration; ++i){
	        if (z_re * z_re + z_im * z_im > 4.f) 
	    	    break;

	        float new_re = z_re * z_re - z_im * z_im;
	 	float new_im = 2.f * z_re * z_im;
		z_re = c_re + new_re;
		z_im = c_im + new_im;
    	    }

	        int* row = (int*)((char*)data + (thisY + a) * pitch);
                row[thisX + b] = i;
	}
    }	
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int size = resX * resY * sizeof(int);
    size_t pitch = 0; 
    int scale = GROUP_SIZE;

    int *d;
    hipHostAlloc(&d, size, hipHostMallocMapped);
    int *data;
    hipMallocPitch(&data, &pitch, resX * sizeof(int), resY);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(resX/threadsPerBlock.x/scale, resY/threadsPerBlock.y/scale);
    mandelKernel<<<numBlocks, threadsPerBlock>>>(data, lowerX, lowerY, stepX, stepY, maxIterations, pitch, scale);

    hipMemcpy2D(d, resX * sizeof(int), data, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, d, size);
    hipFree(data);
    hipHostFree(d);

}
